#include "hip/hip_runtime.h"
#include <nvbench/detail/throw.cuh>
#include <nvbench/nvbench.cuh>

#include <thrust/detail/raw_pointer_cast.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/gather.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/random.h>
#include <thrust/scan.h>
#include <thrust/scatter.h>
#include <thrust/sequence.h>
#include <thrust/shuffle.h>

#include <cub/device/device_memcpy.cuh>
#include <cub/iterator/transform_input_iterator.cuh>

#include <tbm/range_generator.cuh>

#include <cstdint>
#include <limits>
#include <random>
#include <stdexcept>

template <typename ItT, typename T>
void __global__ prnt(ItT data, T num_items)
{
  if (threadIdx.x == 0)
  {
    for (int i = 0; i < num_items; i++)
    {
      auto x = data[i];
      printf("%d: %d\n", i, x);
    }
  }
}

/**
 * @brief Enum class with options for generating the buffer order within memory
 */
enum class buffer_order
{
  // Buffers are randomly shuffled within memory
  RANDOM,

  // Buffer N+1 resides next to buffer N
  CONSECUTIVE
};

/**
 * @brief Function object class template that takes an offset and returns an
 * iterator at the given offset relative to a fixed base iterator.
 *
 * @tparam IteratorT The random-access iterator type to be returned
 */
template <typename IteratorT>
struct offset_to_ptr_op
{
  template <typename T>
  __host__ __device__ __forceinline__ IteratorT operator()(T offset) const
  {
    return base_it + offset;
  }
  IteratorT base_it;
};

/**
 * @brief Used for generating a shuffled but cohesive sequence of output-buffer
 * offsets for the sequence of input-buffers.
 */
template <typename BufferOffsetT,
          typename ByteOffsetT,
          typename BufferSizeItT,
          typename ByteOffsetItT>
void get_shuffled_buffer_offsets(BufferSizeItT buffer_sizes,
                                 BufferOffsetT num_buffers,
                                 ByteOffsetItT shuffled_buffer_offsets,
                                 const unsigned int seed = 320981U)
{
  num_buffers = 100;

  // We're remapping the i-th buffer to pmt_idxs[i]
  thrust::device_vector<BufferOffsetT> pmt_idxs(num_buffers);

  thrust::default_random_engine g{seed};
  auto seq_it = thrust::make_counting_iterator(BufferOffsetT{0});
  thrust::shuffle_copy(seq_it, seq_it + num_buffers, std::begin(pmt_idxs), g);

  // std::cout << " -- Buffer sizes --\n";
  // prnt<<<1, 1>>>(buffer_sizes, num_buffers);

  // std::cout << " -- Permutation indexes --\n";
  // for (int i = 0; i < num_buffers; i++)
  //   std::cout << i << ": " << pmt_idxs[i] << "\n";
  // exit(-1);

  // Compute the offsets using the new mapping
  auto permuted_sizes_it =
    thrust::make_permutation_iterator(buffer_sizes, std::begin(pmt_idxs));
  thrust::device_vector<ByteOffsetT> permuted_offsets(num_buffers);
  thrust::exclusive_scan(permuted_sizes_it,
                         permuted_sizes_it + num_buffers,
                         std::begin(permuted_offsets),
                         static_cast<ByteOffsetT>(0));

  // for (auto permuted_buffer_idx : pmt_idxs)
  // {
  //   permuted_offsets.emplace_back(running_offset);
  //   running_offset += buffer_sizes[permuted_buffer_idx];
  // }

  // Generate the scatter indexes that identify where each buffer was mapped to
  thrust::device_vector<BufferOffsetT> gather_map(num_buffers);
  thrust::scatter(seq_it,
                  seq_it + num_buffers,
                  std::begin(pmt_idxs),
                  std::begin(gather_map));
  // thrust::sequence(permuted_scatter_idxs, permuted_scatter_idxs +
  // num_buffers, BufferOffsetT{0}); for (BufferOffsetT i = 0; i < num_buffers;
  // i++)
  // {
  //   scatter_idxs[pmt_idxs[i]] = i;
  // }

  thrust::device_vector<ByteOffsetT> new_offsets(num_buffers);
  thrust::gather(std::begin(gather_map),
                 std::end(gather_map),
                 std::begin(permuted_offsets),
                 shuffled_buffer_offsets);
  //  std::begin(new_offsets));
  // for (BufferOffsetT i = 0; i < num_buffers; i++)
  // {
  //   new_offsets[i] = permuted_offsets[scatter_idxs[i]];
  // }
}

struct offset_buffers_and_resize_op
{
  std::size_t min_buffer_size{};
  std::size_t max_buffer_size{};
  std::size_t atomic_type_size{};

  template <typename T>
  __device__ __forceinline__ T operator()(const T &val)
  {
    auto rnd_buffer_size = std::abs(val);
    rnd_buffer_size = rnd_buffer_size % (max_buffer_size - min_buffer_size);
    rnd_buffer_size = min_buffer_size + rnd_buffer_size;
    return CUB_ROUND_DOWN_NEAREST(rnd_buffer_size,
                                  static_cast<T>(atomic_type_size));
  }
};

template <typename AtomicT, buffer_order buffer_order>
static void basic(nvbench::state &state,
                  nvbench::type_list<AtomicT, nvbench::enum_type<buffer_order>>)
{
  // Type alias
  using SrcPtrT       = uint8_t *;
  using BufferOffsetT = int32_t;
  using BufferSizeT   = int32_t;
  using ByteOffsetT   = int32_t;

  constexpr auto input_gen  = buffer_order;
  constexpr auto output_gen = buffer_order;

  const auto target_copy_size =
    static_cast<std::size_t>(state.get_int64("Elements"));

  // Make sure buffer ranges are an integer multiple of AtomicT
  const auto min_buffer_size = CUB_ROUND_UP_NEAREST(
    static_cast<std::size_t>(state.get_int64("Min. buffer size")),
    sizeof(AtomicT));
  const auto max_buffer_size = CUB_ROUND_UP_NEAREST(
    static_cast<std::size_t>(state.get_int64("Max. buffer size")),
    sizeof(AtomicT));

  // Skip benchmarks where min. buffer size exceeds max. buffer size
  if (min_buffer_size > max_buffer_size)
  {
    state.skip("Skipping benchmark, as min. buffer size exceeds max. buffer "
               "size.");
    return;
  }

  // Compute number of buffers to generate
  double average_buffer_size = (min_buffer_size + max_buffer_size) / 2.0;
  const auto num_buffers =
    static_cast<std::size_t>(target_copy_size / average_buffer_size);

  // Buffer segment data (their offsets and sizes)
  thrust::device_vector<ByteOffsetT> h_buffer_src_offsets(num_buffers);
  thrust::device_vector<ByteOffsetT> h_buffer_dst_offsets(num_buffers);

  // Generate the buffer sizes
  auto rnd_buffer_sizes =
    tbm::make_range_generator<BufferSizeT,
                              tbm::iterator_style::pointer,
                              tbm::data_pattern::random>(num_buffers);

  // Offset random buffer sizes by the minimum buffer size and make sure buffer
  // sizes are a multiple of the most granular unit (one AtomicT) being copied
  // (round down)
  // prnt<<<1, 1>>>(rnd_buffer_sizes.cbegin(), 100);
  thrust::device_vector<BufferSizeT> buffer_sizes(num_buffers);
  thrust::transform(std::cbegin(rnd_buffer_sizes),
                    std::cend(rnd_buffer_sizes),
                    std::begin(buffer_sizes),
                    offset_buffers_and_resize_op{min_buffer_size,
                                                 max_buffer_size,
                                                 sizeof(AtomicT)});
  // prnt<<<1, 1>>>(thrust::raw_pointer_cast(buffer_sizes.data()), 100);

  // Make sure buffer sizes are a multiple of the most granular unit (one
  // AtomicT) being copied (round down)
  //   for (BufferOffsetT i = 0; i < num_buffers; i++)
  // {
  //   h_buffer_sizes[i] = (h_buffer_sizes[i] / sizeof(AtomicT)) *
  //   sizeof(AtomicT);
  // }

  // For consecutive input buffer order generate the buffer offsets
  if (input_gen == buffer_order::CONSECUTIVE)
  {
    thrust::exclusive_scan(std::begin(buffer_sizes),
                           std::end(buffer_sizes),
                           std::begin(h_buffer_src_offsets),
                           0);
  }
  // For consecutive output buffer order generate the buffer offsets
  if (output_gen == buffer_order::CONSECUTIVE)
  {
    thrust::exclusive_scan(std::begin(buffer_sizes),
                           std::end(buffer_sizes),
                           std::begin(h_buffer_dst_offsets),
                           0);
  }

  // Compute the total bytes to be copied
  ByteOffsetT num_total_bytes = thrust::reduce(std::begin(buffer_sizes),
                                               std::end(buffer_sizes));

  // Shuffle input buffer source-offsets
  unsigned int shuffle_seed = 320981U;
  if (input_gen == buffer_order::RANDOM)
  {
    get_shuffled_buffer_offsets<BufferOffsetT, ByteOffsetT>(
      buffer_sizes.cbegin(),
      static_cast<BufferOffsetT>(buffer_sizes.size()),
      h_buffer_src_offsets.data(),
      shuffle_seed);
    shuffle_seed += 42;
  }

  // Shuffle input buffer source-offsets
  if (output_gen == buffer_order::RANDOM)
  {
    get_shuffled_buffer_offsets<BufferOffsetT, ByteOffsetT>(
      buffer_sizes.cbegin(),
      static_cast<BufferOffsetT>(buffer_sizes.size()),
      h_buffer_dst_offsets.data(),
      shuffle_seed);
  }

  // Get temporary storage requirements
  size_t temp_storage_bytes = 0;
  CubDebugExit(hipcub::DeviceMemcpy::Batched(nullptr,
                                          temp_storage_bytes,
                                          static_cast<SrcPtrT *>(nullptr),
                                          static_cast<SrcPtrT *>(nullptr),
                                          static_cast<BufferSizeT *>(nullptr),
                                          num_buffers));

  // Compute total device memory requirements
  std::size_t total_required_mem = num_total_bytes +                     //
                                   num_total_bytes +                     //
                                   (num_buffers * sizeof(ByteOffsetT)) + //
                                   (num_buffers * sizeof(ByteOffsetT)) + //
                                   (num_buffers * sizeof(BufferSizeT)) + //
                                   temp_storage_bytes;                   //

  // Get available device memory
  std::size_t available_device_mem =
    state.get_device().has_value()
      ? state.get_device().value().get_global_memory_usage().bytes_free
      : 0;

  // Skip benchmark there's insufficient device memory available
  if (available_device_mem < total_required_mem)
  {
    state.skip("Skipping benchmark due to insufficient device memory");
    return;
  }

  thrust::device_vector<uint8_t> d_temp_storage(temp_storage_bytes);

  // Add benchmark reads
  state.add_element_count(num_total_bytes);
  state.add_global_memory_reads<char>(num_total_bytes, "data");
  state.add_global_memory_reads<ByteOffsetT>(num_buffers, "buffer src offsets");
  state.add_global_memory_reads<ByteOffsetT>(num_buffers, "buffer dst offsets");
  state.add_global_memory_reads<BufferSizeT>(num_buffers, "buffer sizes");

  // Add benchmark writes
  state.add_global_memory_writes<char>(num_total_bytes, "data");

  // Prepare random data segment (which serves for the buffer sources)
  thrust::device_vector<uint8_t> d_in_buffer(num_total_bytes);
  thrust::device_vector<uint8_t> d_out_buffer(num_total_bytes);

  // Populate the data source buffer
  thrust::fill(std::begin(d_in_buffer),
               std::end(d_in_buffer),
               std::numeric_limits<uint8_t>::max());

  // Raw pointers into the source and destination buffer
  auto d_in  = thrust::raw_pointer_cast(d_in_buffer.data());
  auto d_out = thrust::raw_pointer_cast(d_out_buffer.data());

  // Prepare device-side data
  thrust::device_vector<ByteOffsetT> d_buffer_src_offsets =
    h_buffer_src_offsets;
  thrust::device_vector<ByteOffsetT> d_buffer_dst_offsets =
    h_buffer_dst_offsets;
  thrust::device_vector<BufferSizeT> d_buffer_sizes = buffer_sizes;

  // Prepare d_buffer_srcs
  offset_to_ptr_op<SrcPtrT> src_transform_op{static_cast<SrcPtrT>(d_in)};
  hipcub::TransformInputIterator<SrcPtrT, offset_to_ptr_op<SrcPtrT>, ByteOffsetT *>
    d_buffer_srcs(thrust::raw_pointer_cast(d_buffer_src_offsets.data()),
                  src_transform_op);

  // Prepare d_buffer_dsts
  offset_to_ptr_op<SrcPtrT> dst_transform_op{static_cast<SrcPtrT>(d_out)};
  hipcub::TransformInputIterator<SrcPtrT, offset_to_ptr_op<SrcPtrT>, ByteOffsetT *>
    d_buffer_dsts(thrust::raw_pointer_cast(d_buffer_dst_offsets.data()),
                  dst_transform_op);

  state.exec([&](nvbench::launch &launch) {
    std::size_t temp_size = d_temp_storage.size(); // need an lvalue
    hipcub::DeviceMemcpy::Batched(thrust::raw_pointer_cast(d_temp_storage.data()),
                               temp_size,
                               d_buffer_srcs,
                               d_buffer_dsts,
                               thrust::raw_pointer_cast(d_buffer_sizes.data()),
                               num_buffers,
                               launch.get_stream());
  });
}

// Column names for type axes:
inline std::vector<std::string> type_axis_names()
{
  return {"AtomicT", "Buffer Order"};
}

// Benchmark for unaligned buffers and buffers aligned to four bytes
using atomic_type = nvbench::type_list<nvbench::uint8_t, nvbench::uint32_t>;

using buffer_orders =
  nvbench::enum_type_list<buffer_order::RANDOM, buffer_order::CONSECUTIVE>;

NVBENCH_DECLARE_ENUM_TYPE_STRINGS(
  buffer_order,
  [](buffer_order data_gen_mode) {
    switch (data_gen_mode)
    {
      case buffer_order::RANDOM:
        return "Random";
      case buffer_order::CONSECUTIVE:
        return "Consecutive";
      default:
        break;
    }
    NVBENCH_THROW(std::runtime_error, "{}", "Unknown data_pattern");
  },
  [](buffer_order data_gen_mode) {
    switch (data_gen_mode)
    {
      case buffer_order::RANDOM:
        return "Buffers are randomly shuffled within memory";
      case buffer_order::CONSECUTIVE:
        return "Consecutive buffers reside cohesively in memory";
      default:
        break;
    }
    NVBENCH_THROW(std::runtime_error, "{}", "Unknown data_pattern");
  })

NVBENCH_BENCH_TYPES(basic, NVBENCH_TYPE_AXES(atomic_type, buffer_orders))
  .set_name("hipcub::DeviceMemcpy::Batched")
  .set_type_axes_names(type_axis_names())
  .add_int64_axis("Min. buffer size", {1, 64 * 1024})
  .add_int64_axis("Max. buffer size", {8, 64, 256, 1024, 64 * 1024})
  .add_int64_power_of_two_axis("Elements", nvbench::range(25, 29, 2));
